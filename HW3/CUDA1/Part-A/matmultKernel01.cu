#include "hip/hip_runtime.h"
///
/// matmultKernel00.cu
/// For COMS E6998 Spring 2023
/// Instructor: Parajit Dube and Kaoutar El Maghraoui
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-01-27
/// Last Modified: 2011-02-23 DVN
///
/// Multiplies two matrices using CUDA: A x B = C
///
/// Copy this file and modify the MatMultKernel device function for
/// each of your experiments. 
///

#include "matmultKernel.h"
#include <stdio.h>

// Define a gpu kernel to perform matrix multiplication
// of A x B = C, but each thread should compute 4 output elements
__global__ void MatMulKernel(Matrix A_mat, Matrix B_mat, Matrix C_mat){
    
    //C_mat.elements[100] = 3.14f;    

  // matrix blocks
  float *A, *B, *C;
  A = A_mat.elements;
  B = B_mat.elements;
  C = C_mat.elements;
  
  // Putting these into registers speeds access.
  int thread_row = threadIdx.y;
  int thread_col = threadIdx.x;
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Each THREAD BLOCK computes one sub matrix Csub of C
  // EACH THREAD creates its own matrix descriptor Csub
  //Csub = &C.elements[C.stride * FOOTPRINT_SIZE * block_row + BLOCK_SIZE * block_col];

  // Each thread computes four elements of Csub in its copy of CValues
  float Cvalue00 = 0;
  float Cvalue01 = 0;
  float Cvalue10 = 0;
  float Cvalue11 = 0;


  // row and column indices within a tile for this thread
  int r0 = thread_row * 2;
  int r1 = r0 + 1;
  int c0 = thread_col * 2;
  int c1 = c0 + 1;

  __shared__ float shared_A[FOOTPRINT_SIZE][FOOTPRINT_SIZE];
  __shared__ float shared_B[FOOTPRINT_SIZE][FOOTPRINT_SIZE];

  // Loop over all sub matrices in block_row of A and block_col of B
  // required to compute Csub. Block multiply each pair of sub matrices
  // and accumulate results
  for (int m = 0;  m < (A_mat.width / FOOTPRINT_SIZE); ++m){


    // each thread will load a 2 x 2 submatrix from global memory into shared memory

    // current tile start points for shared matrix A
    int tile_origin_row_A = block_row * FOOTPRINT_SIZE;
    int tile_origin_col_A = m * FOOTPRINT_SIZE;

    int base_sh_row = thread_row * 2;
    int base_sh_col = thread_col * 2;
    int base_global_row = tile_origin_row_A + base_sh_row;
    int base_global_col = tile_origin_col_A + base_sh_col;
    
    shared_A[base_sh_row + 0][base_sh_col + 0] = A[(base_global_row + 0) * A_mat.stride + (base_global_col + 0)];
    shared_A[base_sh_row + 0][base_sh_col + 1] = A[(base_global_row + 0) * A_mat.stride + (base_global_col + 1)];
    shared_A[base_sh_row + 1][base_sh_col + 0] = A[(base_global_row + 1) * A_mat.stride + (base_global_col + 0)];
    shared_A[base_sh_row + 1][base_sh_col + 1] = A[(base_global_row + 1) * A_mat.stride + (base_global_col + 1)];
    
    // current tile start points for shared matrix B
    int tile_origin_row_B = m * FOOTPRINT_SIZE;
    int tile_origin_col_B = block_col * FOOTPRINT_SIZE;

    base_sh_row = thread_row * 2;
    base_sh_col = thread_col * 2;
    base_global_row = tile_origin_row_B + base_sh_row;
    base_global_col = tile_origin_col_B + base_sh_col;

    shared_B[base_sh_row + 0][base_sh_col + 0] = B[(base_global_row + 0) * B_mat.stride + (base_global_col + 0)];
    shared_B[base_sh_row + 0][base_sh_col + 1] = B[(base_global_row + 0) * B_mat.stride + (base_global_col + 1)];
    shared_B[base_sh_row + 1][base_sh_col + 0] = B[(base_global_row + 1) * B_mat.stride + (base_global_col + 0)];
    shared_B[base_sh_row + 1][base_sh_col + 1] = B[(base_global_row + 1) * B_mat.stride + (base_global_col + 1)];

    
    // Synchronize to ensure all elements are read - at this point all threads have loaded their 2 x 2 submatrices into shared memory
    __syncthreads();

    // Perform required multiplications

    #pragma unroll
    for (int k = 0; k < FOOTPRINT_SIZE; ++k) {
        float A_r0_k = shared_A[r0][k];
        float A_r1_k = shared_A[r1][k];
        float B_k_c0 = shared_B[k][c0];
        float B_k_c1 = shared_B[k][c1];

        Cvalue00 += A_r0_k * B_k_c0;
        Cvalue01 += A_r0_k * B_k_c1;
        Cvalue10 += A_r1_k * B_k_c0;
        Cvalue11 += A_r1_k * B_k_c1;
    }
    // Synchronize threads to ensure all Cvalues are computed before loading the next block
    __syncthreads();
    }

    // Write the computed Cvalue to the output matrix C at the appropriate location

    int global_row = block_row * FOOTPRINT_SIZE + r0;
    int global_col = block_col * FOOTPRINT_SIZE + c0;

    C[(global_row + 0) * C_mat.stride + (global_col + 0)] = Cvalue00;
    C[(global_row + 0) * C_mat.stride + (global_col + 1)] = Cvalue01;
    C[(global_row + 1) * C_mat.stride + (global_col + 0)] = Cvalue10;
    C[(global_row + 1) * C_mat.stride + (global_col + 1)] = Cvalue11;
}
